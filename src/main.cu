#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <gmp.h>
#include <cstdint>
#include <time.h>

#include "cuda/kernels.cuh"

void read_points_from_file(affine_point *points, uint32_t num_points) {

    FILE *file;
    char buffer[130];
    mpz_t big_num;

    // Initialize GMP integer
    mpz_init(big_num);

    // Open the file
    file = fopen("bls12-381_points.txt", "r");
    if (file == NULL) {
        perror("Error opening file");
        return;
    }

    // Read points from the file
    int i = 0;
    while (i < 2 * num_points && fscanf(file, "%s\n", buffer) == 1) {
        mpz_set_str(big_num, buffer, 10); // Read the big number in decimal

        // Extract limbs from the big number
        for (int j = 0; j < TLC; j+=2) {
            mp_limb_t limb = mpz_getlimbn(big_num, j / 2);
            //
            if (i % 2 == 0) {
                points[i / 2].x.limbs[j] = static_cast<uint32_t>(limb & 0xFFFFFFFF);
                points[i / 2].x.limbs[j + 1] = static_cast<uint32_t>((limb >> 32) & 0xFFFFFFFF);

            } else {
                points[i / 2].y.limbs[j] = static_cast<uint32_t>(limb & 0xFFFFFFFF);
                points[i / 2].y.limbs[j + 1] = static_cast<uint32_t>((limb >> 32) & 0xFFFFFFFF);
            }
        }

        i++;
    }

    // Close the file
    fclose(file);
    mpz_clear(big_num);
}


int main() {
	// Sample random points
    int num_points = 1024 * 64;
    int num_points_per_thread = 32;

    // Allocate page-locked memory for points
	affine_point* points;
    hipHostAlloc(&points, num_points * sizeof(affine_point), hipHostMallocDefault);

    // Allocate page-locked memory for results
	affine_point* results;
    hipHostAlloc(&results, (num_points / num_points_per_thread) * sizeof(point_xyzz), hipHostMallocDefault);


    clock_t start = clock();
    read_points_from_file(points, num_points);
    clock_t end = clock();

    double cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Function took %f seconds to execute \n", cpu_time_used);

    hipStream_t memoryStreamHostToDevice, memoryStreamDeviceToHost, runStream;
    hipStreamCreate(&memoryStreamHostToDevice);
    hipStreamCreate(&memoryStreamDeviceToHost);
    hipStreamCreate(&runStream);

    // print_device_properties();

    // init memory
    affine_point *pointsRegionA, *pointsRegionB;
    point_xyzz *resultRegionA, *resultRegionB;

    hipMalloc(&pointsRegionA, sizeof(affine_point) * num_points);
    hipMalloc(&pointsRegionB, sizeof(affine_point) * num_points);
    hipMalloc(&resultRegionA, sizeof(point_xyzz) * (num_points / num_points_per_thread));
    hipMalloc(&resultRegionB, sizeof(point_xyzz) * (num_points / num_points_per_thread));

    printf("Allocated memory\n");

    for (int i = 0; i < 100; i++) {

        // Wait for the GPU to finish
        hipStreamSynchronize(memoryStreamHostToDevice); 
        hipStreamSynchronize(runStream);
        hipStreamSynchronize(memoryStreamDeviceToHost);

        if(i % 2 == 0) {
            hipMemcpyAsync(pointsRegionA, points, sizeof(affine_point) * num_points, hipMemcpyHostToDevice, memoryStreamHostToDevice);
            if(i == 0)
                continue;
            accumulate_kernel<<<64, 32, 0, runStream>>>(resultRegionB, pointsRegionB, num_points);
            hipMemcpyAsync(results, resultRegionA, sizeof(point_xyzz) * (num_points / num_points_per_thread), hipMemcpyDeviceToHost, memoryStreamDeviceToHost);
        } else {
            hipMemcpyAsync(pointsRegionB, points, sizeof(affine_point) * num_points, hipMemcpyHostToDevice, memoryStreamHostToDevice);
            accumulate_kernel<<<64, 32, 0, runStream>>>(resultRegionA, pointsRegionA, num_points);
            if(i == 1)
                continue;
            hipMemcpyAsync(results, resultRegionB, sizeof(point_xyzz) * (num_points / num_points_per_thread), hipMemcpyDeviceToHost, memoryStreamDeviceToHost);
        }


        // Copy result back to host
        //hipMemcpy(ret, retPtrGPU, sizeof(storage) * (num_points / 2), hipMemcpyDeviceToHost);

    }

    // Destroy streams
    hipStreamDestroy(memoryStreamDeviceToHost);
    hipStreamDestroy(memoryStreamHostToDevice);
    hipStreamDestroy(runStream);

    printf("Destroyed streams\n");

    // Free memory 
    hipFree(pointsRegionA);
    hipFree(pointsRegionB);
    hipFree(resultRegionA);
    hipFree(resultRegionB);


    // Free allocated memory
    hipHostFree(points);
    hipHostFree(results);

	return 0;

}